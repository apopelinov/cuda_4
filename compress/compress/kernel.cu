#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include ""
#include "CUDA.h"
#include "Consts.h"

__host__ __device__ void GpuToCompressedSize(int w, int h, int & cw, int & ch)
{
	cw = w / BLOCK_SIZE * COMPRESSED_BLOCK_SIZE;
	ch = h / BLOCK_SIZE * COMPRESSED_BLOCK_SIZE;
}

__device__ void CosineTransform(byte_t src[BLOCK_SIZE][BLOCK_SIZE],
								byte_t dst[COMPRESSED_BLOCK_SIZE][COMPRESSED_BLOCK_SIZE])
{
	int u = threadIdx.x;
	int v = threadIdx.y;
	if (u >= COMPRESSED_BLOCK_SIZE || v >= COMPRESSED_BLOCK_SIZE) return;
	int uu = ZZ[u][v][0], vv = ZZ[u][v][1];
	float uv = 0;
	for (int li = 0; li < BLOCK_SIZE; ++li)
	for (int lj = 0; lj < BLOCK_SIZE; ++lj)
	{
		uv += CC[li][uu] * CC[lj][vv] * src[li][lj] / QY[uu][vv];
	}
	uv = fmaxf(-127.f, fminf(127.f, uv));
	dst[u][v] = (byte_t)((char)uv);
}

__device__ void CosineTransformInv(byte_t dst[BLOCK_SIZE][BLOCK_SIZE],
								   byte_t src[COMPRESSED_BLOCK_SIZE][COMPRESSED_BLOCK_SIZE])
{
	int li = threadIdx.x;
	int lj = threadIdx.y;
	float uv = 0;
	for (int u = 0; u < COMPRESSED_BLOCK_SIZE; ++u)
	for (int v = 0; v < COMPRESSED_BLOCK_SIZE; ++v)
	{
		int uu = ZZ[u][v][0], vv = ZZ[u][v][1];
		float c = (char)src[u][v] * QY[uu][vv];

		uv += CC[li][uu] * CC[lj][vv] * c;
	}
	dst[li][lj] = (byte_t)(fmaxf(0.f, fminf(255.f, uv)));
}

__global__ void DoCompress(const byte_t * src, byte_t * dst, int w, int h)
{
	int cw, ch; GpuToCompressedSize(w, h, cw, ch);

	int bi = blockIdx.x;
	int bj = blockIdx.y;

	int ti = threadIdx.x;
	int tj = threadIdx.y;

	int i = bi * BLOCK_SIZE + ti;
	int j = bj * BLOCK_SIZE + tj;

	int ci = bi * COMPRESSED_BLOCK_SIZE + ti;
	int cj = bj * COMPRESSED_BLOCK_SIZE + tj;

	__shared__ byte_t block[BLOCK_SIZE][BLOCK_SIZE];
	__shared__ byte_t buf[COMPRESSED_BLOCK_SIZE][COMPRESSED_BLOCK_SIZE];

	block[ti][tj] = src[i * w + j];

	__syncthreads();

	CosineTransform(block, buf);

	__syncthreads();

	if (ti < COMPRESSED_BLOCK_SIZE && tj < COMPRESSED_BLOCK_SIZE)
		dst[ci * cw + cj] = buf[ti][tj];
}

__global__ void DoDecompress(const byte_t * src, byte_t * dst, int w, int h)
{
	int cw, ch; GpuToCompressedSize(w, h, cw, ch);

	int bi = blockIdx.x;
	int bj = blockIdx.y;

	int ti = threadIdx.x;
	int tj = threadIdx.y;

	int i = bi * BLOCK_SIZE + ti;
	int j = bj * BLOCK_SIZE + tj;

	int ci = bi * COMPRESSED_BLOCK_SIZE + ti;
	int cj = bj * COMPRESSED_BLOCK_SIZE + tj;

	__shared__ byte_t block[BLOCK_SIZE][BLOCK_SIZE];
	__shared__ byte_t buf[COMPRESSED_BLOCK_SIZE][COMPRESSED_BLOCK_SIZE];

	if (ti < COMPRESSED_BLOCK_SIZE && tj < COMPRESSED_BLOCK_SIZE)
		buf[ti][tj] = src[ci * cw + cj];

	__syncthreads();

	CosineTransformInv(block, buf);

	__syncthreads();

	dst[i * w + j] = block[ti][tj];
}

void StartDeviceCompress(byte_t * src, byte_t * dst, int w, int h)
{
	dim3 threads(BLOCK_SIZE, BLOCK_SIZE);
	dim3 blocks(w / BLOCK_SIZE, h / BLOCK_SIZE);

	int cw, ch; GpuToCompressedSize(w, h, cw, ch);

	byte_t * csrc, * cdst;

	hipMalloc(&csrc, w * h);
	hipMalloc(&cdst, cw * ch);

	hipMemcpy(csrc, src, sizeof(byte_t) * w * h, hipMemcpyHostToDevice);

	DoCompress<<<blocks, threads>>>(csrc, cdst, w, h);

	hipDeviceSynchronize();

	hipMemcpy(dst, cdst, sizeof(byte_t) * cw * ch, hipMemcpyDeviceToHost);

	hipFree(csrc);
	hipFree(cdst);
}

void StartDeviceDecompress(byte_t * dst, byte_t * src, int w, int h)
{
	dim3 threads(BLOCK_SIZE, BLOCK_SIZE);
	dim3 blocks(w / BLOCK_SIZE, h / BLOCK_SIZE);

	int cw, ch; GpuToCompressedSize(w, h, cw, ch);

	byte_t * csrc, *cdst;

	hipMalloc(&csrc, cw * ch);
	hipMalloc(&cdst, w * h);

	hipMemcpy(csrc, src, sizeof(byte_t) * cw * ch, hipMemcpyHostToDevice);

	DoDecompress<<<blocks, threads>>>(csrc, cdst, w, h);

	hipDeviceSynchronize();

	hipMemcpy(dst, cdst, sizeof(byte_t) * w * h, hipMemcpyDeviceToHost);

	hipFree(csrc);
	hipFree(cdst);
}
